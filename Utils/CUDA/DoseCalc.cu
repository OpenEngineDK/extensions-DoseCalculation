#include "hip/hip_runtime.h"
#include <Meta/CUDA.h>

#include <Utils/CUDA/DoseCalc.h>
#include <Utils/CUDA/uint_util.hcu>
#include <Utils/CUDA/DozeCuda.h>

#include <stdlib.h>

typedef unsigned char uchar;
typedef unsigned int  uint;

texture<float, 3, hipReadModeElementType> tex;
uint3 dimensions; // should be placd in constant memory
float3 scale; // should be placed in constant memory

void SetupDoseCalc(unsigned int pbo, 
                   int w, int h, int d, // dimensions
                   float sw, float sh, float sd) // scale
{ 
    cudaGLRegisterBufferObject(pbo);
    CHECK_FOR_CUDA_ERROR();

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    tex.normalized = true;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    hipBindTextureToArray(tex, GetVolumeArray(), channelDesc);

    printf("SetupDoseCalc done: %i\n",pbo);

    CHECK_FOR_CUDA_ERROR();
    dimensions = make_uint3(w, h, d);
    scale = make_float3(sw, sh, sd);
}

__device__ unsigned int GetRadiologicalDepth(float3 coordinate, float3 source, float3 dimensions, float3 scale){
    // The vector from the coordinate to the source
    float3 vec = source - coordinate;

    float dist = length(vec);

    // Instead of alpha between [0; 1] use the length of the
    // vector. (and in the future scale the length to make it match
    // texcoords?)

    // delta.x is the distance the beam has to travel between crossing
    // zy-planes.
    float3 delta = dist * scale / vec;

    float3 texCoords = coordinate / dimensions;

    // The border texcoords (@TODO: Doesn't have to be calculated for
    // every voxel, move outside later.)
    float3 border = make_float3((vec.x > 0) ? 1 : 0,
                                (vec.y > 0) ? 1 : 0,
                                (vec.z > 0) ? 1 : 0);

    // The remaining distance to the next crossing.
    float3 alpha;

    while (alpha.x != border.x ||
           alpha.y != border.y ||
           alpha.z != border.z){
        
    }

    return 0;
}

__global__ void radioDepth(float* output, uint3 dims, float3 scale, Beam beam) {
    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    // lookup via tex3D(...);

    uint3 coordinate = idx_to_co(idx, dims);

    output[idx] = (coordinate.x / dims.x + coordinate.y / dims.y + coordinate.z / dims.z) * 0.25f;
}

__global__ void doseCalc(uint *d_output) {

}

void RunDoseCalc(unsigned int pbo, Beam beam, int beamlet_x, int beamlet_y, float dx, float dy, float dz) {
    // Map the buffer object that we want to write the radiological depth to.
    float* radiologicalDepth;
    cudaGLMapBufferObject( (void**)&radiologicalDepth, pbo);

    dim3 blockDim(512,1,1);
    double entries = dimensions.x * dimensions.y * dimensions.z;
	dim3 gridDim((uint)(ceil(entries/blockDim.x)), 1, 1);

    radioDepth<<< gridDim, blockDim >>>(radiologicalDepth, 
                                        dimensions,
                                        scale,
                                        beam);

    cudaGLUnmapBufferObject(pbo);    
}
