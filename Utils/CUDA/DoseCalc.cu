#include "hip/hip_runtime.h"
#include <Meta/CUDA.h>

#include <Utils/CUDA/DoseCalc.h>
#include <Utils/CUDA/uint_util.hcu>
#include <Utils/CUDA/float_util.hcu>
#include <Utils/CUDA/DozeCuda.h>

#include <stdlib.h>

struct Matrix3x3 {
    float3 e[3];

    __host__ __device__ Matrix3x3(){
        e[0] = make_float3(1, 0, 0);
        e[1] = make_float3(0, 1, 0);
        e[2] = make_float3(0, 0, 1);
    }

    __host__ __device__ float3 mul(float3 m){
        return make_float3(dot(m, e[0]),
                           dot(m, e[1]),
                           dot(m, e[2]));
    }

    __host__ __device__ Matrix3x3 getInverse(){
        Matrix3x3 res;

        float e0112 = e[0].y * e[1].z;
        float e0122 = e[0].y * e[2].z;
        float e0211 = e[0].z * e[1].y;
        float e0221 = e[0].z * e[2].y;
        float e1021 = e[1].x * e[2].y;
        float e1022 = e[1].x * e[2].z;
        float e1122 = e[1].y * e[2].z;
        float e1221 = e[1].z * e[2].y;
        float e1120 = e[1].y * e[2].x;
        float e1220 = e[1].z * e[2].x;

        float determinant = e[0].x * (e1122 - e1221) - e[0].y * (e1022 - e1220) + e[0].z * (e1021 - e1120);
        float invDet = 1.0f / determinant;
        
        res.e[0].x = (e1122 - e1221) * invDet;
        res.e[0].y = (e0221 - e0122) * invDet;
        res.e[0].z = (e0112 - e0211) * invDet;

        res.e[1].x = (e1220 - e1022) * invDet;
        res.e[1].y = (e[0].x * e[2].z - e[0].z * e[2].x) * invDet;
        res.e[1].z = (e[0].z * e[1].x - e[0].x * e[1].z) * invDet;

        res.e[2].x = (e1021 - e1120) * invDet;
        res.e[2].y = (e[0].y * e[2].x - e[0].x * e[2].y) * invDet;
        res.e[2].z = (e[0].x * e[1].y - e[0].y * e[1].x) * invDet;

        return res;
    }

    void print(){
        printf("[[%f, %f, %f]\n", e[0].x, e[0].y, e[0].z);
        printf("[%f, %f, %f]\n", e[1].x, e[1].y, e[1].z);
        printf("[%f, %f, %f]]\n", e[2].x, e[2].y, e[2].z);
    }
};

struct CudaBeam {
    float3 source;
    Matrix3x3 invCone1;
    Matrix3x3 invCone2;
};

typedef unsigned char uchar;
typedef unsigned int  uint;

texture<float, 3, hipReadModeElementType> tex;
uint3 dimensions; // should be placed in constant memory along with their inverse
float3 scale; // should be placed in constant memory

void SetupDoseCalc(float** cuDoseArr, 
                   int w, int h, int d, // dimensions
                   float sw, float sh, float sd) // scale
{ 
    
    hipMalloc((void**)cuDoseArr, sizeof(float)*w*h*d);
    CHECK_FOR_CUDA_ERROR();

    printf("malloc: %d,%d,%d\n",w,h,d);

    // Setup texture
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(tex, GetVolumeArray(), channelDesc);

    printf("SetupDoseCalc done\n");

    CHECK_FOR_CUDA_ERROR();
    dimensions = make_uint3(w, h, d);
    scale = make_float3(sw, sh, sd);
}

__device__ bool VoxelInsiceBeam(Matrix3x3 invCone1, Matrix3x3 invCone2, float3 point){
    return (invCone1.mul(point) >= 0.0f && invCone2.mul(point) >= 0.0f);
}

__device__ float GetRadiologicalDepth(uint3 coordinate, float3 source, uint3 dimensions, float3 scale){
    // The vector from the coordinate to the source
    const float3 vec = source - coordinate;

    const float dist = length(vec);

    // Instead of alpha between [0; 1] use the length of the
    // vector. This is usefull for when we need the length travelede
    // when accumulating radiological depth.

    // delta.x is the distance the beam has to travel between crossing
    // zy-planes.
    const float delta[3] = {dist * scale.x / vec.x,
                            dist * scale.y / vec.y,
                            dist * scale.z / vec.z};

    const int texDelta[3] = {(vec.x > 0) ? 1 : -1,
                             (vec.y > 0) ? 1 : -1,
                             (vec.z > 0) ? 1 : -1};

    // The border texcoords (@TODO: Doesn't have to be calculated for
    // every voxel, move outside later.)
    const int border[3] = {(vec.x > 0) ? dimensions.x : -1,
                           (vec.y > 0) ? dimensions.y : -1,
                           (vec.z > 0) ? dimensions.z : -1};
    
    // The remaining distance to the next crossing.
    //float3 alpha = delta;
    float alpha[3] = {delta[0], delta[1], delta[2]};

    int texCoord[3] = {coordinate.x, coordinate.y, coordinate.z};

    const int maxItr = 100;

    float radiologicalDepth = 0;
    int itr = 0;

    while ((texCoord[0] != border[0] &&
            texCoord[1] != border[1] &&
            texCoord[2] != border[2]) &&
           /*0 <= texCoord[0] && texCoord[0] < dimensions.x &&
           0 <= texCoord[1] && texCoord[1] < dimensions.y &&
           0 <= texCoord[2] && texCoord[2] < dimensions.z && */
           itr < maxItr){
        itr++;

        // is x less then y?
        int minIndex = (alpha[0] < alpha[1]) ? 0 : 1;
        // is the above min less then z?
        minIndex = (alpha[minIndex] < alpha[2]) ? minIndex : 2;

        // We need to store the smallest alpha value so we can advance
        // the alpha with that value.
        float advance = alpha[minIndex];

        // Add the delta value of the crossing dimension to prepare
        // for the next crossing.
        alpha[minIndex] += delta[minIndex];

        // Advance the alpha values.
        alpha[0] -= advance;
        alpha[1] -= advance;
        alpha[2] -= advance;

        // Advance the texture coordinates
        texCoord[minIndex] += texDelta[minIndex];

        // Add the radiological length for this step to the overall
        // depth.
        radiologicalDepth = advance * tex3D(tex, texCoord[0], texCoord[1], texCoord[2]);
    }

    return radiologicalDepth;
}

__global__ void radioDepth(float* output, uint3 dims, float3 scale, float3 source) {
    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    const uint3 coordinate = idx_to_co(idx, dims);

    float rDepth = GetRadiologicalDepth(coordinate, source, dims, scale);

    if (idx < dims.x * dims.y * dims.z)
        output[idx] = rDepth;
}

__global__ void doseCalc(uint *d_output) {
    
}

void RunDoseCalc(float* cuDoseArr, Beam beam, int beamlet_x, int beamlet_y, float dx, float dy, float dz) {
    float3 source = make_float3(beam.src[0], beam.src[1], beam.src[2]);

    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(dimensions.x * dimensions.z / blockSize.x, dimensions.y / blockSize.y);

    /*
    radioDepth<<< gridSize, blockSize >>>((float*)cuDoseArr, 
                                        dimensions,
                                        scale,
                                        source);
    */

    CHECK_FOR_CUDA_ERROR();
    printf("Hurray\n");

    Matrix3x3 id = Matrix3x3();
    id.e[0].y = 1;
    id.e[2].x = 5;
    id = id.getInverse();
    id.print();
}
