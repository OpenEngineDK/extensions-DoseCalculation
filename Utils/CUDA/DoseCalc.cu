#include "hip/hip_runtime.h"
#include <Meta/CUDA.h>

#include <Utils/CUDA/DoseCalc.h>
#include <Utils/CUDA/uint_util.hcu>
#include <Utils/CUDA/DozeCuda.h>

#include <stdlib.h>

typedef unsigned char uchar;
typedef unsigned int  uint;

texture<float, 3, hipReadModeElementType> tex;
uint3 dimensions; // should be placd in constant memory
float3 scale; // should be placed in constant memory

void SetupDoseCalc(float** cuDoseArr, 
                   int w, int h, int d, // dimensions
                   float sw, float sh, float sd) // scale
{ 
    
    hipMalloc((void**)cuDoseArr, sizeof(float)*w*h*d);
    CHECK_FOR_CUDA_ERROR();

    printf("malloc: %d,%d,%d\n",w,h,d);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    hipBindTextureToArray(tex, GetVolumeArray(), channelDesc);

    printf("SetupDoseCalc done\n");

    CHECK_FOR_CUDA_ERROR();
    dimensions = make_uint3(w, h, d);
    scale = make_float3(sw, sh, sd);
}

__device__ float GetRadiologicalDepth(uint3 coordinate, float3 source, uint3 dimensions, float3 scale){
    // The vector from the coordinate to the source
    const float3 vec = source - coordinate;

    const float dist = length(vec);

    // Instead of alpha between [0; 1] use the length of the
    // vector. This is usefull for when we need the length travelede
    // when accumulating radiological depth.

    // delta.x is the distance the beam has to travel between crossing
    // zy-planes.
    const float delta[3] = {dist * scale.x / vec.x,
                      dist * scale.y / vec.y,
                      dist * scale.z / vec.z};

    int texCoord[3] = {coordinate.x, coordinate.y, coordinate.z};

    const int texDelta[3] = {(vec.x > 0) ? 1 : -1,
                       (vec.y > 0) ? 1 : -1,
                       (vec.z > 0) ? 1 : -1};

    // The border texcoords (@TODO: Doesn't have to be calculated for
    // every voxel, move outside later.)
    const int border[3] = {(vec.x > 0) ? dimensions.x : -1,
                           (vec.y > 0) ? dimensions.y : -1,
                           (vec.z > 0) ? dimensions.z : -1};
    
    // The remaining distance to the next crossing.
    //float3 alpha = delta;
    float alpha[3] = {delta[0], delta[1], delta[2]};

    const int maxItr = 10;

    float radiologicalDepth = 0;
    int itr = 0;

    while (itr < maxItr){
        itr++;

        // is x less then y?
        int minIndex = (alpha[0] < alpha[1]) ? alpha[0] : alpha[1];
        // is the above min less then z?
        minIndex = (minIndex < alpha[2]) ? minIndex : alpha[2];

        // We need to store the smallest alpha value so we can advance
        // the alpha with that value.
        float advance = alpha[minIndex];

        // Add the delta value of the crossing dimension to prepare
        // for the next crossing.
        alpha[minIndex] += delta[minIndex];

        // Advance the alpha values.
        alpha[0] -= advance;
        alpha[1] -= advance;
        alpha[2] -= advance;

        // Advance the texture coordinates
        texCoord[minIndex] += texDelta[minIndex];

        // Add the radiological length for this step to the overall
        // depth.
        radiologicalDepth = tex3D(tex, texCoord[0], texCoord[1], texCoord[2]);
    }

    /*
    while (texCoord[0] != border.x ||
           texCoord[1] != border.y ||
           texCoord[2] != border.z ||
           itr < maxItr){
    
        itr++;

        // is x less then y?
        int minIndex = (alpha[0] < alpha[1]) ? alpha[0] : alpha[1];
        // is the above min less then z?
        minIndex = (minIndex < alpha[2]) ? minIndex : alpha[2];
        
        // We need to store the smallest alpha value so we can advance
        // the alpha with that value.
        float advance = alpha[minIndex];

        // Add the delta value of the crossing dimension to prepare
        // for the next crossing.
        alpha[minIndex] += delta[minIndex];

        // Advance the alpha values.
        alpha[0] -= advance;
        alpha[1] -= advance;
        alpha[2] -= advance;

        // Advance the texture coordinates
        texCoord[minIndex] += texDelta[minIndex];

        // Add the radiological length for this step to the overall
        // depth.
        radiologicalDepth = tex3D(tex, texCoord[0], texCoord[1], texCoord[2]);
    }
    */
    
    return radiologicalDepth;
}

__global__ void radioDepth(float* output, uint3 dims, float3 scale, float3 source) {
    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    const uint3 coordinate = idx_to_co(idx, dims);

    float rDepth = GetRadiologicalDepth(coordinate, source, dims, scale);

    output[idx] = (float(coordinate.x) / float(dims.x)); // + coordinate.y / dims.y + coordinate.z / dims.z) * 0.25f;
}

__global__ void doseCalc(uint *d_output) {

}

void RunDoseCalc(float* cuDoseArr, Beam beam, int beamlet_x, int beamlet_y, float dx, float dy, float dz) {
    float3 source = make_float3(beam.src[0], beam.src[1], beam.src[2]);

    /*
    const unsigned int blockDimX = 512;
    const dim3 blockSize(blockDimX,1,1);
    const float entries = dimensions.x * dimensions.y * dimensions.z;
    const dim3 gridSize(ceil(entries/(float)blockDimX), 1, 1);
    */

    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(dimensions.x * dimensions.z / blockSize.x, dimensions.y / blockSize.y);

    radioDepth<<< gridSize, blockSize >>>((float*)cuDoseArr, 
                                        dimensions,
                                        scale,
                                        source);
    CHECK_FOR_CUDA_ERROR();
    printf("Hurray\n");

}
