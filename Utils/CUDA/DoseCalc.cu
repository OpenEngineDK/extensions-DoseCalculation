#include "hip/hip_runtime.h"
#include <Meta/CUDA.h>

#include <Utils/CUDA/DoseCalc.h>
#include <Utils/CUDA/uint_util.hcu>
#include <Utils/CUDA/float_util.hcu>
#include <Utils/CUDA/DozeCuda.h>

#include <stdlib.h>

#include <Utils/CUDA/Matrix3x3.h>

struct CudaBeam {
    float3 src;
    Matrix3x3 cone1;
    Matrix3x3 invCone1;
    Matrix3x3 cone2;
    Matrix3x3 invCone2;

    __host__ void operator() (Beam b){
        src.x = b.src[0];
        src.y = b.src[1];
        src.z = b.src[2];

        cone1(b.p1 - b.src, b.p2 - b.src, b.p3 - b.src);
        invCone1 = cone1.getInverse();

        cone2(b.p1 - b.src, b.p4 - b.src, b.p3 - b.src);
        invCone2 = cone2.getInverse();
    }
};

typedef unsigned char uchar;
typedef unsigned int  uint;

texture<float, 3, hipReadModeElementType> tex;

uint3 dimensions;
__constant__ uint3 dims;
__constant__ float3 scale;
__constant__ CudaBeam beam;

void SetupDoseCalc(float** cuDoseArr, 
                   int w, int h, int d, // dimensions
                   float sw, float sh, float sd) // scale
{ 
    hipMalloc((void**)cuDoseArr, sizeof(float)*w*h*d);
    CHECK_FOR_CUDA_ERROR();

    // Setup texture
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(tex, GetVolumeArray(), channelDesc);
    CHECK_FOR_CUDA_ERROR();

    printf("Dimmensions: %d,%d,%d\n",w,h,d);
    dimensions = make_uint3(w, h, d);
    hipMemcpyToSymbol(HIP_SYMBOL(dims), &dimensions, sizeof(uint3));
    CHECK_FOR_CUDA_ERROR();

    printf("Scale: %f,%f,%f\n", sw, sh, sd);
    hipMemcpyToSymbol(HIP_SYMBOL(scale), &make_float3(sw, sh, sd), sizeof(float3));
    CHECK_FOR_CUDA_ERROR();

    printf("SetupDoseCalc done\n");
}

__device__ bool VoxelInsideBeam(float3 point){
    // __constant__ CudaBeam beam
    float3 translatedPoint = point - beam.src;
    return beam.invCone1.mul(translatedPoint) >= 0
        || beam.invCone2.mul(translatedPoint) >= 0;
}

__device__ float GetRadiologicalDepth(const uint3 textureCoord){
    // __constant__ uint3 dims
    // __constant__ float3 scale
    // __constant__ CudaBeam beam;

    // The texture coordinate is in buffer space. Not yet scaled.

    // Coordinate in world space.
    const float3 coordinate = make_float3(textureCoord.x * scale.x, 
                                          textureCoord.y * scale.y, 
                                          textureCoord.z * scale.z);

    // The vector from the coordinate to the source
    const float3 vec = beam.src - coordinate;

    const float dist = length(vec);

    // delta.x is the distance the beam has to travel between crossing
    // zy-planes. The distance is always positive.
    const float delta[3] = {abs(dist / vec.x),
                            abs(dist / vec.y),
                            abs(dist / vec.z)};
    
    const int texDelta[3] = {(vec.x > 0) ? 1 : -1,
                             (vec.y > 0) ? 1 : -1,
                             (vec.z > 0) ? 1 : -1};

    // The border texcoords (@TODO: Doesn't have to be calculated for
    // every voxel, make __constant__ and update before each run.)
    const int border[3] = {(vec.x > 0) ? dims.x : -1,
                           (vec.y > 0) ? dims.y : -1,
                           (vec.z > 0) ? dims.z : -1};
    
    // The remaining distance to the next crossing.
    float alpha[3] = {delta[0], delta[1], delta[2]};

    int texCoord[3] = {textureCoord.x, textureCoord.y, textureCoord.z};

    float radiologicalDepth = 0;

    while (0 <= texCoord[0] && texCoord[0] < dims.x &&
           0 <= texCoord[1] && texCoord[1] < dims.y &&
           0 <= texCoord[2] && texCoord[2] < dims.z
           /*texCoord[0] != border[0] &&
           texCoord[1] != border[1] &&
           texCoord[2] != border[2]*/){
        
        // is x less then y?
        int minIndex = (alpha[0] < alpha[1]) ? 0 : 1;
        // is the above min less then z?
        minIndex = (alpha[minIndex] < alpha[2]) ? minIndex : 2;

        // We need to store the smallest alpha value so we can advance
        // the alpha with that value.
        float advance = alpha[minIndex];

        // Add the delta value of the crossing dimension to prepare
        // for the next crossing.
        alpha[minIndex] += delta[minIndex];

        // Advance the alpha values.
        alpha[0] -= advance;
        alpha[1] -= advance;
        alpha[2] -= advance;

        // Advance the texture coordinates
        texCoord[minIndex] += texDelta[minIndex];

        // Add the radiological length for this step to the overall
        // depth.
        radiologicalDepth += advance * tex3D(tex, texCoord[0], texCoord[1], texCoord[2]);
    }

    return radiologicalDepth;
}

/**
 * Calculates the radiological depth of each voxel and stores it in
 * the output array.
 */
__global__ void radioDepth(float* output) {
    // __constant__ uint3 dims
    // __constant__ float3 scale
    // __constant__ CudaBeam beam;

    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    const uint3 coordinate = idx_to_co(idx, dims);
   
    float rDepth = GetRadiologicalDepth(coordinate);

    if (idx < dims.x * dims.y * dims.z)
        output[idx] = rDepth;
}

/**
 * Calculates for each voxel wether it is inside the beam or not.
 *
 * param output An array of the voxels interest. Contains 1.0 if the
 * voxel is in the beam otherwise 0.0f.
 */
__global__ void voxelsOfInterest(float* output) {
    // __constant__ uint3 dims
    // __constant__ float3 scale
    // __constant__ CudaBeam beam;

    const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    const uint3 coordinate = idx_to_co(idx, dims);

    // @todo multiply by scale or divide?
    const float3 fcoord = make_float3(coordinate.x * scale.x,
                                      coordinate.y * scale.y,
                                      coordinate.z * scale.z);
   
    if (idx < dims.x * dims.y * dims.z)
        output[idx] = (VoxelInsideBeam(fcoord)) ? 1.0f : 0.0f;
}

/**
 * Calculate the score of each beamlet, dependent on the voxels it hits.
 *
 * param input An array of radiological depths for each voxel.
 * param output An boolean array of how each beamlet performed.
 */
__global__ void doseCalc(float* input, uint *output) {
    // __constant__ uint3 dims
    // __constant__ float3 scale
    // __constant__ CudaBeam beam;

    // Calculate the inverse matrix of the beams 2 convex cones.

    // For each plane calculate wether the beam hits and in which
    // voxels it does.

    // Then rate the beam based on each voxel it hits.
}

void RunDoseCalc(float* cuDoseArr, Beam oeBeam, int beamlet_x, int beamlet_y, int kernel) {
    CudaBeam _beam;
    _beam(oeBeam);

    hipMemcpyToSymbol(HIP_SYMBOL(beam), &_beam, sizeof(CudaBeam));
    CHECK_FOR_CUDA_ERROR();

    /* const dim3 blockSize(16, 16, 1); */
    /* const dim3 gridSize(dimensions.x * dimensions.z / blockSize.x, dimensions.y / blockSize.y); */

    const dim3 blockSize(512, 1, 1);
    const dim3 gridSize(dimensions.x * dimensions.z * dimensions.y / blockSize.x, 1);

    switch(kernel){
    case 0:
        radioDepth<<< gridSize, blockSize >>>(cuDoseArr);
        break;
    case 1:
        voxelsOfInterest<<< gridSize, blockSize >>>(cuDoseArr);
        break;
    default:
        voxelsOfInterest<<< gridSize, blockSize >>>(cuDoseArr);
    }

    /*
      // Voxel of interest debug print.
    printf("Source\n");
    printf("[%f, %f, %f]\n", _beam.src.x, _beam.src.y, _beam.src.z);

    printf("\nCone 1\n");
    _beam.cone1.print();

    printf("\nCone 1 inverse\n");
    _beam.invCone1.print();

    printf("\nCone 1 inverse * ((0, 0, 0) - source)\n");
    float3 res = _beam.invCone1.mul(make_float3(0.0f) - _beam.src);
    printf("[%f, %f, %f]\n", res.x, res.y, res.z);

    printf("\nCone 2\n");
    _beam.cone2.print();

    printf("\nCone 2 inverse * ((0, 0, 0) - source)\n");
    res = _beam.invCone2.mul(make_float3(0.0f) - _beam.src);
    printf("[%f, %f, %f]\n", res.x, res.y, res.z);
    */

    CHECK_FOR_CUDA_ERROR();
}
