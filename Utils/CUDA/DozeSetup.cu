#include <Meta/CUDA.h>
#include <Utils/CUDA/Doze.h>
// Lets do Teh Setup!


hipArray *d_volumeArray = 0;

texture<float, 3, hipReadModeElementType> tex;

hipArray* GetVolumeArray() {
    return d_volumeArray;
}

void SetupDoze(const float* data, int w, int h, int d) {
    
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();    
    hipExtent ext = make_hipExtent(w,h,d);


    hipMalloc3DArray(&d_volumeArray, &channelDesc, ext);
    CHECK_FOR_CUDA_ERROR();

    hipMemcpy3DParms copyParams = {0};
    
    copyParams.srcPtr = make_hipPitchedPtr((void*)data, 
                                            ext.width*sizeof(float),
                                            ext.width,
                                            ext.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent = ext;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    CHECK_FOR_CUDA_ERROR();
    
    tex.normalized = true;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    hipBindTextureToArray(tex, d_volumeArray, channelDesc);
    CHECK_FOR_CUDA_ERROR();

    printf("Doze Are SETUP!\n");
    
}
