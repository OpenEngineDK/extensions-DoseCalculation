#include "hip/hip_runtime.h"
#include "RayCaster.h"
#include <Utils/CUDA/DozeCuda.h>
#include <Utils/CUDA/uint_util.hcu>
#include <Meta/CUDA.h>


struct Matrix4x4 {
    float4 e[4]; // rows
    
    // i == row, j == col
    __device__ float get(uint i, uint j) {
        switch (j) {
        case 0: 
            return e[i].x;
        case 1: 
            return e[i].y;       
        case 2: 
            return e[i].z;
        case 3: 
            return e[i].w;
        }
        return 0.0f;
    }

    __device__ float4 mul(float4 v) {
        float4 r;
        
        r.x = dot(v,e[0]);
        r.y = dot(v,e[1]);
        r.z = dot(v,e[2]);
        r.w = dot(v,e[3]);

        return r;
    }
};

struct Ray {
	float3 origin;
	float3 direction;
};


__constant__ Matrix4x4 c_invViewMatrix;

texture<float, 3, hipReadModeElementType> tex;

uint3 dimensions1;
float3 scale1;

void SetupRayCaster(int pbo,  const float* data,
                    int w, int h, int d,
                    float sw, float sh, float sd ) {
    
    cudaGLRegisterBufferObject(pbo);
    CHECK_FOR_CUDA_ERROR();

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    
    
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    hipBindTextureToArray(tex, GetVolumeArray(), channelDesc);
    CHECK_FOR_CUDA_ERROR();

    dimensions1 = make_uint3(w, h, d);
    scale1 = make_float3(sw, sh, sd);
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.direction;
    float3 tbot = invR * (boxmin - r.origin);
    float3 ttop = invR * (boxmax - r.origin);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}


__device__ Ray RayForPoint(uint u, uint v, uint width, uint height,float pm00, float pm11) {
    //float x = ((2*u - width) / float(width));
    //float y = ((2*v - height) / float(height));


    float x = (u / float(width)) * 2.0f-1.0f ;
    float y = (v / float(height)) *2.0f-1.0f;

    float4 projPoint = make_float4(x,y,-1,0);
    projPoint.x = x / pm00;
    projPoint.y = y / pm11;
    
    float4 rDir4 = c_invViewMatrix.mul(projPoint); // c_invViewMatrix is transposed!

    float3 rDir = make_float3(rDir4);
    
    
    Ray rr;
    rr.origin.x = c_invViewMatrix.get(0,3);
    rr.origin.y = c_invViewMatrix.get(1,3);
    rr.origin.z = c_invViewMatrix.get(2,3);

    rr.direction = normalize(rDir);

    return rr;
}

__global__ void rayCaster(uint *d_output, float* d_intense, uint imageW, uint imageH,
                          float minIt, float maxIt,
                          float transferOffset, float transferScale,
                          float pm00, float pm11,
                          uint3 dims,
                          float3 scale) {
    int maxD = dims.x;
    float tStep = 1.0f;
    
    float4 col = make_float4(0.0f);
    
    float3 boxMin = make_float3(0.0f);
    float3 boxMax = make_float3( dims.x*scale.x, 
                                 dims.y*scale.y,
                                 dims.z*scale.z);


    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    Ray r = RayForPoint(x,y,imageW,imageH,pm00,pm11);
    // We got the ray now, lets intersect it with the box..
    
    float tnear, tfar;
	int hit = intersectBox(r, boxMin, boxMax, &tnear, &tfar);
        
    //float3 inversedd = make_float3(1.0f, 1.0f, 1.0f) / dd;

    if (hit) {
        //if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
        //col.x = 1.0f;
        float t = tnear;
        //float t = tfar;

        float3 p1 = r.origin + r.direction*tnear;
        float3 p2 = r.origin + r.direction*tfar;
        float3 dp = p2-p1;
        float dist = sqrt(dp.x*dp.x + dp.y*dp.y + dp.z*dp.z);
        maxD = dist;
        for (int i=0;i<maxD;i++) {                        
            float3 pos = r.origin + r.direction*t;
            t += tStep;

            // descale it
            float3 spos = pos / scale;// * inversedd;
            //pos = spos;
            
            if (i > 0 &&(spos.x < 0 ||
                spos.y < 0 ||
                spos.z < 0 ||
                spos.x > dims.x ||
                spos.y > dims.y ||
                spos.z > dims.z
                 )) {
                break;
            }

            float sample = tex3D(tex, spos.x, spos.y, spos.z);
            if (sample > minIt && sample <= maxIt) {

                col = make_float4(sample);

                /* float inte = 1.0f; */
                /* uint3 posi = make_uint3(spos); */
                /* int idx = co_to_idx(posi, dims); */
                
                /* if (idx < dims.x*dims.y*dims.z) {  */
                /*     col.y = 0.0f;  */
                /* } */
                /*     col.y = 0.0; */
                /*     //inte = d_intense[idx]; */
                /* } else { */

                /* } */
                                

                //col.x = inte;

                break;
            }


        }
    }


    // Insert directly in loop instead of break.

    // Make the size of the pbo big enough that we can't write outside
    // it. W're doing the calculations anyways. Might as well discard
    // them later and not slowdown every calculation.

    //if ((x < imageW) && (y < imageH)) {
        // write output color
    uint i = __umul24(y, imageW) + x;
    d_output[i] = rgbaFloatToInt(col);
        ///}

    
}

void RenderToPBO(int pbo, float* cuDoseArr, int width, int height, float* invMat, float pm00, float pm11, float minIt, float maxIt) {
    hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invMat, sizeof(float4)*4);
    CHECK_FOR_CUDA_ERROR();

 
    uint* p;
    cudaGLMapBufferObject((void**)&p,pbo);
    CHECK_FOR_CUDA_ERROR();


    
    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(width / blockSize.x, height / blockSize.y);

    float3 po = make_float3(100,100,30);
    uint3 poi = make_uint3(po);
    int idx = co_to_idx(poi,dimensions1);
    //printf("[%d] %d,%d,%d\n",idx,poi.x,poi.y,poi.z);
    //printf(" %d\n",dimensions1.x*dimensions1.y*dimensions1.z);
    
    //printf("cast: %d,%d,%d\n",dimensions1.x,dimensions1.y,dimensions1.z);
    rayCaster<<<gridSize, blockSize>>>(p,cuDoseArr,width,height,
                                       minIt,maxIt,1,1,pm00,pm11,dimensions1,scale1);
    CHECK_FOR_CUDA_ERROR();

    cudaGLUnmapBufferObject(pbo);

    CHECK_FOR_CUDA_ERROR();

}
