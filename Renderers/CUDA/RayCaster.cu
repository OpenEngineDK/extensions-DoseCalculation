#include "hip/hip_runtime.h"
#include "RayCaster.h"
#include <Meta/CUDA.h>

typedef unsigned char uchar;

texture<uchar, 3, hipReadModeNormalizedFloat> tex;
hipArray *d_volumeArray = 0;



void SetupRayCaster(int pbo, const float* data, int w, int h, int d) {
    
    cudaGLRegisterBufferObject(pbo);
    CHECK_FOR_CUDA_ERROR();


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    
    hipExtent ext = make_hipExtent(h,w,d);

    hipMalloc3DArray(&d_volumeArray, &channelDesc, ext);
    CHECK_FOR_CUDA_ERROR();

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)data, 
                                            ext.width*sizeof(float),
                                            ext.width,
                                            ext.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent = ext;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    CHECK_FOR_CUDA_ERROR();
    
    tex.normalized = true;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.addressMode[2] = hipAddressModeWrap;

    hipBindTextureToArray(tex, d_volumeArray, channelDesc);
    CHECK_FOR_CUDA_ERROR();

}

__global__ void rayCaster(uint* p_out, uint imageW, uint imageH) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;
    // read from 3D texture
    float voxel = tex3D(tex, u, v, 0.0f);

    if ((x < imageW) && (y < imageH)) {
        // write output color
        uint i = __umul24(y, imageW) + x;
        p_out[i] = voxel*255;
        
    }

}

void RenderToPBO(int pbo, int width, int height) {
    uint* p;
    cudaGLMapBufferObject((void**)&p,pbo);
    CHECK_FOR_CUDA_ERROR();
    
    const dim3 blockSize(16, 16, 1);
    const dim3 gridSize(width / blockSize.x, height / blockSize.y);


    rayCaster<<<gridSize, blockSize>>>(p,width,height);

    CHECK_FOR_CUDA_ERROR();

    cudaGLUnmapBufferObject(pbo);
    CHECK_FOR_CUDA_ERROR();

}
